#include "hip/hip_runtime.h"
#ifndef KERNEL
#define KERNEL

#include <constants.cu>
#include <math.h>
#include <stdio.h>

// Code modified from
// https://github.com/uber/h3/tree/e0aae450ffa7a63a3b7982573c88325b42231332 with
// license https://github.com/uber/h3/blob/master/LICENSE Apache 2.0

// Apache License
// Version 2.0, January 2004
// http://www.apache.org/licenses/

// TERMS AND CONDITIONS FOR USE, REPRODUCTION, AND DISTRIBUTION

// 1. Definitions.

// "License" shall mean the terms and conditions for use, reproduction,
// and distribution as defined by Sections 1 through 9 of this document.

// "Licensor" shall mean the copyright owner or entity authorized by
// the copyright owner that is granting the License.

// "Legal Entity" shall mean the union of the acting entity and all
// other entities that control, are controlled by, or are under common
// control with that entity. For the purposes of this definition,
// "control" means (i) the power, direct or indirect, to cause the
// direction or management of such entity, whether by contract or
// otherwise, or (ii) ownership of fifty percent (50%) or more of the
// outstanding shares, or (iii) beneficial ownership of such entity.

// "You" (or "Your") shall mean an individual or Legal Entity
// exercising permissions granted by this License.

// "Source" form shall mean the preferred form for making modifications,
// including but not limited to software source code, documentation
// source, and configuration files.

// "Object" form shall mean any form resulting from mechanical
// transformation or translation of a Source form, including but
// not limited to compiled object code, generated documentation,
// and conversions to other media types.

// "Work" shall mean the work of authorship, whether in Source or
// Object form, made available under the License, as indicated by a
// copyright notice that is included in or attached to the work
// (an example is provided in the Appendix below).

// "Derivative Works" shall mean any work, whether in Source or Object
// form, that is based on (or derived from) the Work and for which the
// editorial revisions, annotations, elaborations, or other modifications
// represent, as a whole, an original work of authorship. For the purposes
// of this License, Derivative Works shall not include works that remain
// separable from, or merely link (or bind by name) to the interfaces of,
// the Work and Derivative Works thereof.

// "Contribution" shall mean any work of authorship, including
// the original version of the Work and any modifications or additions
// to that Work or Derivative Works thereof, that is intentionally
// submitted to Licensor for inclusion in the Work by the copyright owner
// or by an individual or Legal Entity authorized to submit on behalf of
// the copyright owner. For the purposes of this definition, "submitted"
// means any form of electronic, verbal, or written communication sent
// to the Licensor or its representatives, including but not limited to
// communication on electronic mailing lists, source code control systems,
// and issue tracking systems that are managed by, or on behalf of, the
// Licensor for the purpose of discussing and improving the Work, but
// excluding communication that is conspicuously marked or otherwise
// designated in writing by the copyright owner as "Not a Contribution."

// "Contributor" shall mean Licensor and any individual or Legal Entity
// on behalf of whom a Contribution has been received by Licensor and
// subsequently incorporated within the Work.

// 2. Grant of Copyright License. Subject to the terms and conditions of
// this License, each Contributor hereby grants to You a perpetual,
// worldwide, non-exclusive, no-charge, royalty-free, irrevocable
// copyright license to reproduce, prepare Derivative Works of,
// publicly display, publicly perform, sublicense, and distribute the
// Work and such Derivative Works in Source or Object form.

// 3. Grant of Patent License. Subject to the terms and conditions of
// this License, each Contributor hereby grants to You a perpetual,
// worldwide, non-exclusive, no-charge, royalty-free, irrevocable
// (except as stated in this section) patent license to make, have made,
// use, offer to sell, sell, import, and otherwise transfer the Work,
// where such license applies only to those patent claims licensable
// by such Contributor that are necessarily infringed by their
// Contribution(s) alone or by combination of their Contribution(s)
// with the Work to which such Contribution(s) was submitted. If You
// institute patent litigation against any entity (including a
// cross-claim or counterclaim in a lawsuit) alleging that the Work
// or a Contribution incorporated within the Work constitutes direct
// or contributory patent infringement, then any patent licenses
// granted to You under this License for that Work shall terminate
// as of the date such litigation is filed.

// 4. Redistribution. You may reproduce and distribute copies of the
// Work or Derivative Works thereof in any medium, with or without
// modifications, and in Source or Object form, provided that You
// meet the following conditions:

// (a) You must give any other recipients of the Work or
// Derivative Works a copy of this License; and

// (b) You must cause any modified files to carry prominent notices
// stating that You changed the files; and

// (c) You must retain, in the Source form of any Derivative Works
// that You distribute, all copyright, patent, trademark, and
// attribution notices from the Source form of the Work,
// excluding those notices that do not pertain to any part of
// the Derivative Works; and

// (d) If the Work includes a "NOTICE" text file as part of its
// distribution, then any Derivative Works that You distribute must
// include a readable copy of the attribution notices contained
// within such NOTICE file, excluding those notices that do not
// pertain to any part of the Derivative Works, in at least one
// of the following places: within a NOTICE text file distributed
// as part of the Derivative Works; within the Source form or
// documentation, if provided along with the Derivative Works; or,
// within a display generated by the Derivative Works, if and
// wherever such third-party notices normally appear. The contents
// of the NOTICE file are for informational purposes only and
// do not modify the License. You may add Your own attribution
// notices within Derivative Works that You distribute, alongside
// or as an addendum to the NOTICE text from the Work, provided
// that such additional attribution notices cannot be construed
// as modifying the License.

// You may add Your own copyright statement to Your modifications and
// may provide additional or different license terms and conditions
// for use, reproduction, or distribution of Your modifications, or
// for any such Derivative Works as a whole, provided Your use,
// reproduction, and distribution of the Work otherwise complies with
// the conditions stated in this License.

// 5. Submission of Contributions. Unless You explicitly state otherwise,
// any Contribution intentionally submitted for inclusion in the Work
// by You to the Licensor shall be under the terms and conditions of
// this License, without any additional terms or conditions.
// Notwithstanding the above, nothing herein shall supersede or modify
// the terms of any separate license agreement you may have executed
// with Licensor regarding such Contributions.

// 6. Trademarks. This License does not grant permission to use the trade
// names, trademarks, service marks, or product names of the Licensor,
// except as required for reasonable and customary use in describing the
// origin of the Work and reproducing the content of the NOTICE file.

// 7. Disclaimer of Warranty. Unless required by applicable law or
// agreed to in writing, Licensor provides the Work (and each
// Contributor provides its Contributions) on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied, including, without limitation, any warranties or conditions
// of TITLE, NON-INFRINGEMENT, MERCHANTABILITY, or FITNESS FOR A
// PARTICULAR PURPOSE. You are solely responsible for determining the
// appropriateness of using or redistributing the Work and assume any
// risks associated with Your exercise of permissions under this License.

// 8. Limitation of Liability. In no event and under no legal theory,
// whether in tort (including negligence), contract, or otherwise,
// unless required by applicable law (such as deliberate and grossly
// negligent acts) or agreed to in writing, shall any Contributor be
// liable to You for damages, including any direct, indirect, special,
// incidental, or consequential damages of any character arising as a
// result of this License or out of the use or inability to use the
// Work (including but not limited to damages for loss of goodwill,
// work stoppage, computer failure or malfunction, or any and all
// other commercial damages or losses), even if such Contributor
// has been advised of the possibility of such damages.

// 9. Accepting Warranty or Additional Liability. While redistributing
// the Work or Derivative Works thereof, You may choose to offer,
// and charge a fee for, acceptance of support, warranty, indemnity,
// or other liability obligations and/or rights consistent with this
// License. However, in accepting such obligations, You may act only
// on Your own behalf and on Your sole responsibility, not on behalf
// of any other Contributor, and only if You agree to indemnify,
// defend, and hold each Contributor harmless for any liability
// incurred by, or claims asserted against, such Contributor by reason
// of your accepting any such warranty or additional liability.

// END OF TERMS AND CONDITIONS

// APPENDIX: How to apply the Apache License to your work.

// To apply the Apache License to your work, attach the following
// boilerplate notice, with the fields enclosed by brackets "[]"
// replaced with your own identifying information. (Don't include
// the brackets!)  The text should be enclosed in the appropriate
// comment syntax for the file format. We also recommend that a
// file or class name and description of purpose be included on the
// same "printed page" as the copyright notice for easier
// identification within third-party archives.

// Copyright [yyyy] [name of copyright owner]

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

// http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/**
 * Convert from decimal degrees to radians.
 *
 * @param degrees The decimal degrees.
 * @return The corresponding radians.
 */
__device__ double degsToRads(double degrees) { return degrees * M_PI_180; }

/**
 * Normalizes radians to a value between 0.0 and two PI.
 *
 * @param rads The input radians value.
 * @return The normalized radians value.
 */
__device__ double _posAngleRads(double rads) {
  double tmp = ((rads < 0.0) ? rads + M_2PI : rads);
  if (rads >= M_2PI)
    tmp -= M_2PI;
  return tmp;
}
/**
 * Determines the azimuth to p2 from p1 in radians.
 *
 * @param p1 The first spherical coordinates.
 * @param p2 The second spherical coordinates.
 * @return The azimuth in radians from p1 to p2.
 */
__device__ double _geoAzimuthRads(const LatLng *p1, const LatLng *p2) {
  return atan2(cos(p2->lat) * sin(p2->lng - p1->lng),
               cos(p1->lat) * sin(p2->lat) -
                   sin(p1->lat) * cos(p2->lat) * cos(p2->lng - p1->lng));
}

/**
 * Returns whether or not a resolution is a Class III grid. Note that odd
 * resolutions are Class III and even resolutions are Class II.
 * @param res The H3 resolution.
 * @return 1 if the resolution is a Class III grid, and 0 if the resolution is
 *         a Class II grid.
 */
__device__ int isResolutionClassIII(int res) { return res % 2; }

/**
 * Calculate the 3D coordinate on unit sphere from the latitude and longitude.
 *
 * @param geo The latitude and longitude of the point.
 * @param v The 3D coordinate of the point.
 */
__device__ void _geoToVec3d(const LatLng *geo, Vec3d *v) {
  double r = cos(geo->lat);

  v->z = sin(geo->lat);
  v->x = cos(geo->lng) * r;
  v->y = sin(geo->lng) * r;
}
/**
 * Square of a number
 *
 * @param x The input number.
 * @return The square of the input number.
 */
__device__ double _square(double x) { return x * x; }

/**
 * Calculate the square of the distance between two 3D coordinates.
 *
 * @param v1 The first 3D coordinate.
 * @param v2 The second 3D coordinate.
 * @return The square of the distance between the given points.
 */
__device__ double _pointSquareDist(const Vec3d *v1, const Vec3d *v2) {
  return _square(v1->x - v2->x) + _square(v1->y - v2->y) +
         _square(v1->z - v2->z);
}

/**
 * Encodes a coordinate on the sphere to the corresponding icosahedral face and
 * containing 2D hex coordinates relative to that face center.
 *
 * @param g The spherical coordinates to encode.
 * @param res The desired H3 resolution for the encoding.
 * @param face The icosahedral face containing the spherical coordinates.
 * @param v The 2D hex coordinates of the cell containing the point.
 */
__device__ void _geoToHex2d(const LatLng *g, int res, int *face, Vec2d *v) {
  Vec3d v3d;
  _geoToVec3d(g, &v3d);

  // determine the icosahedron face
  *face = 0;
  double sqd = _pointSquareDist(&faceCenterPoint[0], &v3d);
  for (int f = 1; f < NUM_ICOSA_FACES; f++) {
    double sqdT = _pointSquareDist(&faceCenterPoint[f], &v3d);
    if (sqdT < sqd) {
      *face = f;
      sqd = sqdT;
    }
  }

  // cos(r) = 1 - 2 * sin^2(r/2) = 1 - 2 * (sqd / 4) = 1 - sqd/2
  double r = acos(1 - sqd / 2);

  if (r < EPSILON) {
    v->x = v->y = 0.0;
    return;
  }

  // now have face and r, now find CCW theta from CII i-axis
  double theta =
      _posAngleRads(faceAxesAzRadsCII[*face][0] -
                    _posAngleRads(_geoAzimuthRads(&faceCenterGeo[*face], g)));

  // adjust theta for Class III (odd resolutions)
  if (isResolutionClassIII(res))
    theta = _posAngleRads(theta - M_AP7_ROT_RADS);

  // perform gnomonic scaling of r
  r = tan(r);

  // scale for current resolution length u
  r /= RES0_U_GNOMONIC;
  for (int i = 0; i < res; i++)
    r *= M_SQRT7;

  // we now have (r, theta) in hex2d with theta ccw from x-axes

  // convert to local x,y
  v->x = r * cos(theta);
  v->y = r * sin(theta);
}

/**
 * Normalizes ijk coordinates by setting the components to the smallest possible
 * values. Works in place.
 *
 * @param c The ijk coordinates to normalize.
 */
__device__ void _ijkNormalize(CoordIJK *c) {
  // remove any negative values
  if (c->i < 0) {
    c->j -= c->i;
    c->k -= c->i;
    c->i = 0;
  }

  if (c->j < 0) {
    c->i -= c->j;
    c->k -= c->j;
    c->j = 0;
  }

  if (c->k < 0) {
    c->i -= c->k;
    c->j -= c->k;
    c->k = 0;
  }

  // remove the min value if needed
  int min = c->i;
  if (c->j < min)
    min = c->j;
  if (c->k < min)
    min = c->k;
  if (min > 0) {
    c->i -= min;
    c->j -= min;
    c->k -= min;
  }
}

/**
 * Determine the containing hex in ijk+ coordinates for a 2D cartesian
 * coordinate vector (from DGGRID).
 *
 * @param v The 2D cartesian coordinate vector.
 * @param h The ijk+ coordinates of the containing hex.
 */
__device__ void _hex2dToCoordIJK(const Vec2d *v, CoordIJK *h) {
  double a1, a2;
  double x1, x2;
  int m1, m2;
  double r1, r2;

  // quantize into the ij system and then normalize
  h->k = 0;

  a1 = fabs(v->x); // These used to be fabsl calls, but undefined on device
  a2 = fabs(v->y);

  // first do a reverse conversion
  x2 = a2 / M_SIN60;
  x1 = a1 + x2 / 2.0;

  // check if we have the center of a hex
  m1 = x1;
  m2 = x2;

  // otherwise round correctly
  r1 = x1 - m1;
  r2 = x2 - m2;

  if (r1 < 0.5) {
    if (r1 < 1.0 / 3.0) {
      if (r2 < (1.0 + r1) / 2.0) {
        h->i = m1;
        h->j = m2;
      } else {
        h->i = m1;
        h->j = m2 + 1;
      }
    } else {
      if (r2 < (1.0 - r1)) {
        h->j = m2;
      } else {
        h->j = m2 + 1;
      }

      if ((1.0 - r1) <= r2 && r2 < (2.0 * r1)) {
        h->i = m1 + 1;
      } else {
        h->i = m1;
      }
    }
  } else {
    if (r1 < 2.0 / 3.0) {
      if (r2 < (1.0 - r1)) {
        h->j = m2;
      } else {
        h->j = m2 + 1;
      }

      if ((2.0 * r1 - 1.0) < r2 && r2 < (1.0 - r1)) {
        h->i = m1;
      } else {
        h->i = m1 + 1;
      }
    } else {
      if (r2 < (r1 / 2.0)) {
        h->i = m1 + 1;
        h->j = m2;
      } else {
        h->i = m1 + 1;
        h->j = m2 + 1;
      }
    }
  }

  // now fold across the axes if necessary

  if (v->x < 0.0) {
    if ((h->j % 2) == 0) // even
    {
      long long int axisi = h->j / 2;
      long long int diff = h->i - axisi;
      h->i = h->i - 2.0 * diff;
    } else {
      long long int axisi = (h->j + 1) / 2;
      long long int diff = h->i - axisi;
      h->i = h->i - (2.0 * diff + 1);
    }
  }

  if (v->y < 0.0) {
    h->i = h->i - (2 * h->j + 1) / 2;
    h->j = -1 * h->j;
  }

  _ijkNormalize(h);
}

/** @brief Find base cell given FaceIJK.
 *
 * Given the face number and a resolution 0 ijk+ coordinate in that face's
 * face-centered ijk coordinate system, return the base cell located at that
 * coordinate.
 *
 * Valid ijk+ lookup coordinates are from (0, 0, 0) to (2, 2, 2).
 */
__device__ int _faceIjkToBaseCell(const FaceIJK *h) {
  return faceIjkBaseCells[h->face][h->coord.i][h->coord.j][h->coord.k].baseCell;
}

/**
 * Encodes a coordinate on the sphere to the FaceIJK address of the containing
 * cell at the specified resolution.
 *
 * @param g The spherical coordinates to encode.
 * @param res The desired H3 resolution for the encoding.
 * @param h The FaceIJK address of the containing cell at resolution res.
 */
__device__ void _geoToFaceIjk(const LatLng *g, int res, FaceIJK *h) {
  // first convert to hex2d
  Vec2d v;
  _geoToHex2d(g, res, &h->face, &v);

  // then convert to ijk+
  _hex2dToCoordIJK(&v, &h->coord);
}

/**
 * Add two ijk coordinates.
 *
 * @param h1 The first set of ijk coordinates.
 * @param h2 The second set of ijk coordinates.
 * @param sum The sum of the two sets of ijk coordinates.
 */
__device__ void _ijkAdd(const CoordIJK *h1, const CoordIJK *h2, CoordIJK *sum) {
  sum->i = h1->i + h2->i;
  sum->j = h1->j + h2->j;
  sum->k = h1->k + h2->k;
}

/**
 * Subtract two ijk coordinates.
 *
 * @param h1 The first set of ijk coordinates.
 * @param h2 The second set of ijk coordinates.
 * @param diff The difference of the two sets of ijk coordinates (h1 - h2).
 */
__device__ void _ijkSub(const CoordIJK *h1, const CoordIJK *h2,
                        CoordIJK *diff) {
  diff->i = h1->i - h2->i;
  diff->j = h1->j - h2->j;
  diff->k = h1->k - h2->k;
}

/**
 * Uniformly scale ijk coordinates by a scalar. Works in place.
 *
 * @param c The ijk coordinates to scale.
 * @param factor The scaling factor.
 */
__device__ void _ijkScale(CoordIJK *c, int factor) {
  c->i *= factor;
  c->j *= factor;
  c->k *= factor;
}

/**
 * Returns whether or not two ijk coordinates contain exactly the same
 * component values.
 *
 * @param c1 The first set of ijk coordinates.
 * @param c2 The second set of ijk coordinates.
 * @return 1 if the two addresses match, 0 if they do not.
 */
__device__ int _ijkMatches(const CoordIJK *c1, const CoordIJK *c2) {
  return (c1->i == c2->i && c1->j == c2->j && c1->k == c2->k);
}

/**
 * Find the normalized ijk coordinates of the indexing parent of a cell in a
 * counter-clockwise aperture 7 grid. Works in place.
 *
 * @param ijk The ijk coordinates.
 */
__device__ void _upAp7(CoordIJK *ijk) {
  // convert to CoordIJ
  int i = ijk->i - ijk->k;
  int j = ijk->j - ijk->k;

  ijk->i = (int)round((3 * i - j) / 7.0);
  ijk->j = (int)round((i + 2 * j) / 7.0);
  ijk->k = 0;
  _ijkNormalize(ijk);
}

/**
 * Find the normalized ijk coordinates of the indexing parent of a cell in a
 * clockwise aperture 7 grid. Works in place.
 *
 * @param ijk The ijk coordinates.
 */
__device__ void _upAp7r(CoordIJK *ijk) {
  // convert to CoordIJ
  int i = ijk->i - ijk->k;
  int j = ijk->j - ijk->k;

  ijk->i = (int)round((2 * i + j) / 7.0);
  ijk->j = (int)round((3 * j - i) / 7.0);
  ijk->k = 0;
  _ijkNormalize(ijk);
}

/**
 * Find the normalized ijk coordinates of the hex centered on the indicated
 * hex at the next finer aperture 7 counter-clockwise resolution. Works in
 * place.
 *
 * @param ijk The ijk coordinates.
 */
__device__ void _downAp7(CoordIJK *ijk) {
  // res r unit vectors in res r+1
  CoordIJK iVec = {3, 0, 1};
  CoordIJK jVec = {1, 3, 0};
  CoordIJK kVec = {0, 1, 3};

  _ijkScale(&iVec, ijk->i);
  _ijkScale(&jVec, ijk->j);
  _ijkScale(&kVec, ijk->k);

  _ijkAdd(&iVec, &jVec, ijk);
  _ijkAdd(ijk, &kVec, ijk);

  _ijkNormalize(ijk);
}

/**
 * Find the normalized ijk coordinates of the hex centered on the indicated
 * hex at the next finer aperture 7 clockwise resolution. Works in place.
 *
 * @param ijk The ijk coordinates.
 */
__device__ void _downAp7r(CoordIJK *ijk) {
  // res r unit vectors in res r+1
  CoordIJK iVec = {3, 1, 0};
  CoordIJK jVec = {0, 3, 1};
  CoordIJK kVec = {1, 0, 3};

  _ijkScale(&iVec, ijk->i);
  _ijkScale(&jVec, ijk->j);
  _ijkScale(&kVec, ijk->k);

  _ijkAdd(&iVec, &jVec, ijk);
  _ijkAdd(ijk, &kVec, ijk);

  _ijkNormalize(ijk);
}

// /**
//  * Determines the H3 digit corresponding to a unit vector in ijk coordinates.
//  *
//  * @param ijk The ijk coordinates; must be a unit vector.
//  * @return The H3 digit (0-6) corresponding to the ijk unit vector, or
//  * INVALID_DIGIT on failure.
//  */
// __device__ Direction _unitIjkToDigit(const CoordIJK *ijk) {
//     CoordIJK c = *ijk;
//     _ijkNormalize(&c);

//     Direction digit = INVALID_DIGIT;
//     for (Direction i = CENTER_DIGIT; i < NUM_DIGITS; i++) {
//         if (_ijkMatches(&c, &UNIT_VECS[i])) {
//             digit = i;
//             break;
//         }
//     }

//     return digit;
// }

/**
 * Determines the H3 digit corresponding to a unit vector in ijk coordinates.
 *
 * @param ijk The ijk coordinates; must be a unit vector.
 * @return The H3 digit (0-6) corresponding to the ijk unit vector, or
 * INVALID_DIGIT on failure.
 */
__device__ Direction _unitIjkToDigit(const CoordIJK *ijk) {
  CoordIJK c = *ijk;
  _ijkNormalize(&c);

  // TODO: Replicate enum loop
  // CENTER_DIGIT = 0,
  Direction digit = CENTER_DIGIT;
  if (_ijkMatches(&c, &UNIT_VECS[digit])) {
    return digit;
  }
  // K_AXES_DIGIT = 1,
  digit = K_AXES_DIGIT;
  if (_ijkMatches(&c, &UNIT_VECS[digit])) {
    return digit;
  }
  // J_AXES_DIGIT = 2,
  digit = J_AXES_DIGIT;
  if (_ijkMatches(&c, &UNIT_VECS[digit])) {
    return digit;
  }
  // JK_AXES_DIGIT = J_AXES_DIGIT | K_AXES_DIGIT, /* 3 */
  digit = JK_AXES_DIGIT;
  if (_ijkMatches(&c, &UNIT_VECS[digit])) {
    return digit;
  }
  // I_AXES_DIGIT = 4,
  digit = I_AXES_DIGIT;
  if (_ijkMatches(&c, &UNIT_VECS[digit])) {
    return digit;
  }
  // IK_AXES_DIGIT = I_AXES_DIGIT | K_AXES_DIGIT, /* 5 */
  digit = IK_AXES_DIGIT;
  if (_ijkMatches(&c, &UNIT_VECS[digit])) {
    return digit;
  }
  // IJ_AXES_DIGIT = I_AXES_DIGIT | J_AXES_DIGIT, /* 6 */
  digit = IJ_AXES_DIGIT;
  if (_ijkMatches(&c, &UNIT_VECS[digit])) {
    return digit;
  }

  return INVALID_DIGIT;
}

/** @brief Find base cell given FaceIJK.
 *
 * Given the face number and a resolution 0 ijk+ coordinate in that face's
 * face-centered ijk coordinate system, return the number of 60' ccw rotations
 * to rotate into the coordinate system of the base cell at that coordinates.
 *
 * Valid ijk+ lookup coordinates are from (0, 0, 0) to (2, 2, 2).
 */
__device__ int _faceIjkToBaseCellCCWrot60(const FaceIJK *h) {
  return faceIjkBaseCells[h->face][h->coord.i][h->coord.j][h->coord.k].ccwRot60;
}

/** @brief Return whether or not the indicated base cell is a pentagon. */
__device__ int _isBaseCellPentagon(int baseCell) {
  if (baseCell < 0 || baseCell >= NUM_BASE_CELLS) {
    // Base cells less than zero can not be represented in an index
    return false;
  }
  return baseCellData[baseCell].isPentagon;
}

/**
 * Returns the highest resolution non-zero digit in an H3Index.
 * @param h The H3Index.
 * @return The highest resolution non-zero digit in the H3Index.
 */
__device__ Direction _h3LeadingNonZeroDigit(H3Index h) {
  for (int r = 1; r <= H3_GET_RESOLUTION(h); r++)
    if (H3_GET_INDEX_DIGIT(h, r))
      return H3_GET_INDEX_DIGIT(h, r);

  // if we're here it's all 0's
  return CENTER_DIGIT;
}

/** @brief Return whether or not the tested face is a cw offset face.
 */
__device__ bool _baseCellIsCwOffset(int baseCell, int testFace) {
  return baseCellData[baseCell].cwOffsetPent[0] == testFace ||
         baseCellData[baseCell].cwOffsetPent[1] == testFace;
}

/**
 * Rotates indexing digit 60 degrees counter-clockwise. Returns result.
 *
 * @param digit Indexing digit (between 1 and 6 inclusive)
 */
__device__ Direction _rotate60ccw(Direction digit) {
  switch (digit) {
  case K_AXES_DIGIT:
    return IK_AXES_DIGIT;
  case IK_AXES_DIGIT:
    return I_AXES_DIGIT;
  case I_AXES_DIGIT:
    return IJ_AXES_DIGIT;
  case IJ_AXES_DIGIT:
    return J_AXES_DIGIT;
  case J_AXES_DIGIT:
    return JK_AXES_DIGIT;
  case JK_AXES_DIGIT:
    return K_AXES_DIGIT;
  default:
    return digit;
  }
}
/**
 * Rotates indexing digit 60 degrees clockwise. Returns result.
 *
 * @param digit Indexing digit (between 1 and 6 inclusive)
 */
__device__ Direction _rotate60cw(Direction digit) {
  switch (digit) {
  case K_AXES_DIGIT:
    return JK_AXES_DIGIT;
  case JK_AXES_DIGIT:
    return J_AXES_DIGIT;
  case J_AXES_DIGIT:
    return IJ_AXES_DIGIT;
  case IJ_AXES_DIGIT:
    return I_AXES_DIGIT;
  case I_AXES_DIGIT:
    return IK_AXES_DIGIT;
  case IK_AXES_DIGIT:
    return K_AXES_DIGIT;
  default:
    return digit;
  }
}

/**
 * Rotate an H3Index 60 degrees counter-clockwise.
 * @param h The H3Index.
 */
__device__ H3Index _h3Rotate60ccw(H3Index h) {
  for (int r = 1, res = H3_GET_RESOLUTION(h); r <= res; r++) {
    Direction oldDigit = H3_GET_INDEX_DIGIT(h, r);
    H3_SET_INDEX_DIGIT(h, r, _rotate60ccw(oldDigit));
  }

  return h;
}

/**
 * Rotate an H3Index 60 degrees clockwise.
 * @param h The H3Index.
 */
__device__ H3Index _h3Rotate60cw(H3Index h) {
  for (int r = 1, res = H3_GET_RESOLUTION(h); r <= res; r++) {
    H3_SET_INDEX_DIGIT(h, r, _rotate60cw(H3_GET_INDEX_DIGIT(h, r)));
  }

  return h;
}

/**
 * Rotate an H3Index 60 degrees counter-clockwise about a pentagonal center.
 * @param h The H3Index.
 */
__device__ H3Index _h3RotatePent60ccw(H3Index h) {
  // rotate in place; skips any leading 1 digits (k-axis)

  int foundFirstNonZeroDigit = 0;
  for (int r = 1, res = H3_GET_RESOLUTION(h); r <= res; r++) {
    // rotate this digit
    H3_SET_INDEX_DIGIT(h, r, _rotate60ccw(H3_GET_INDEX_DIGIT(h, r)));

    // look for the first non-zero digit so we
    // can adjust for deleted k-axes sequence
    // if necessary
    if (!foundFirstNonZeroDigit && H3_GET_INDEX_DIGIT(h, r) != 0) {
      foundFirstNonZeroDigit = 1;

      // adjust for deleted k-axes sequence
      if (_h3LeadingNonZeroDigit(h) == K_AXES_DIGIT)
        h = _h3Rotate60ccw(h);
    }
  }
  return h;
}

/**
 * Convert an FaceIJK address to the corresponding H3Index.
 * @param fijk The FaceIJK address.
 * @param res The cell resolution.
 * @return The encoded H3Index (or H3_NULL on failure).
 */
__device__ H3Index _faceIjkToH3(const FaceIJK *fijk, int res) {
  // initialize the index
  H3Index h = H3_INIT;
  H3_SET_MODE(h, H3_CELL_MODE);
  H3_SET_RESOLUTION(h, res);

  // check for res 0/base cell
  if (res == 0) {
    if (fijk->coord.i > MAX_FACE_COORD || fijk->coord.j > MAX_FACE_COORD ||
        fijk->coord.k > MAX_FACE_COORD) {
      // out of range input
      return H3_NULL;
    }

    H3_SET_BASE_CELL(h, _faceIjkToBaseCell(fijk));
    return h;
  }

  // we need to find the correct base cell FaceIJK for this H3 index;
  // start with the passed in face and resolution res ijk coordinates
  // in that face's coordinate system
  FaceIJK fijkBC = *fijk;

  // return h;
  // build the H3Index from finest res up
  // adjust r for the fact that the res 0 base cell offsets the indexing
  // digits
  CoordIJK *ijk = &fijkBC.coord;
  for (int r = res - 1; r >= 0; r--) {
    CoordIJK lastIJK = *ijk;
    CoordIJK lastCenter;
    if (isResolutionClassIII(r + 1)) {
      // rotate ccw
      _upAp7(ijk);
      lastCenter = *ijk;
      _downAp7(&lastCenter);
    } else {
      // rotate cw
      _upAp7r(ijk);
      lastCenter = *ijk;
      _downAp7r(&lastCenter);
    }

    CoordIJK diff;
    _ijkSub(&lastIJK, &lastCenter, &diff);
    _ijkNormalize(&diff);

    H3_SET_INDEX_DIGIT(h, r + 1, _unitIjkToDigit(&diff));
  }

  // fijkBC should now hold the IJK of the base cell in the
  // coordinate system of the current face

  if (fijkBC.coord.i > MAX_FACE_COORD || fijkBC.coord.j > MAX_FACE_COORD ||
      fijkBC.coord.k > MAX_FACE_COORD) {
    // out of range input
    return H3_NULL;
  }

  // lookup the correct base cell
  int baseCell = _faceIjkToBaseCell(&fijkBC);
  H3_SET_BASE_CELL(h, baseCell);

  // rotate if necessary to get canonical base cell orientation
  // for this base cell
  int numRots = _faceIjkToBaseCellCCWrot60(&fijkBC);
  if (_isBaseCellPentagon(baseCell)) {
    // force rotation out of missing k-axes sub-sequence
    if (_h3LeadingNonZeroDigit(h) == K_AXES_DIGIT) {
      // check for a cw/ccw offset face; default is ccw
      if (_baseCellIsCwOffset(baseCell, fijkBC.face)) {
        h = _h3Rotate60cw(h);
      } else {
        h = _h3Rotate60ccw(h);
      }
    }

    for (int i = 0; i < numRots; i++)
      h = _h3RotatePent60ccw(h);
  } else {
    for (int i = 0; i < numRots; i++) {
      h = _h3Rotate60ccw(h);
    }
  }

  return h;
}

/**
 * Encodes a coordinate on the sphere to the H3 index of the containing cell at
 * the specified resolution.
 *
 * Returns 0 on invalid input.
 *
 * @param g The spherical coordinates to encode.
 * @param res The desired H3 resolution for the encoding.
 * @param out The encoded H3Index.
 * @returns E_SUCCESS (0) on success, another value otherwise
 */
__device__ H3Error latLngToCell(const LatLng *g, int res, H3Index *out) {
  if (res < 0 || res > MAX_H3_RES) {
    return E_RES_DOMAIN;
  }
  if (!isfinite(g->lat) || !isfinite(g->lng)) {
    return E_LATLNG_DOMAIN;
  }

  FaceIJK fijk;
  _geoToFaceIjk(g, res, &fijk);
  *out = _faceIjkToH3(&fijk, res);
  if (*out) {
    return E_SUCCESS;
  } else {
    return E_FAILED;
  }
}

void __global__ kernel_h3(float *lat_device, float *lon_device,
                          uint64_t *idx_device, int length, int resolution) {
  int gid = threadIdx.x + blockDim.x * blockIdx.x;

  LatLng location;
  H3Index indexed;
  while (gid < length) {

    location.lat = degsToRads(lat_device[gid]);
    location.lng = degsToRads(lon_device[gid]);

    if (latLngToCell(&location, resolution, &indexed) != E_SUCCESS) {
      idx_device[gid] = H3_NULL;
    } else {
      idx_device[gid] = indexed;
    }
    gid += blockDim.x * gridDim.x;
  }
  __syncthreads();
}

void __global__ kernel_h3_unified(float *data_device, uint64_t *idx_device,
                                  int length, int resolution) {
  int gid = threadIdx.x + blockDim.x * blockIdx.x;

  LatLng location;
  H3Index indexed;
  while (gid < length) {
    int idx = gid * 2;
    location.lat = degsToRads(data_device[idx]);
    location.lng = degsToRads(data_device[idx + 1]);

    if (latLngToCell(&location, resolution, &indexed) != E_SUCCESS) {
      idx_device[gid] = H3_NULL;
    } else {
      idx_device[gid] = indexed;
    }
    gid += blockDim.x * gridDim.x;
  }
  __syncthreads();
}

#endif