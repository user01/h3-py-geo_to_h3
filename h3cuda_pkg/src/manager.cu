#include <assert.h>
#include <inttypes.h>
#include <iostream>
#include <kernel.cu>
#include <manager.hh>
#include <stdio.h>
#include <utility.cu>
using namespace std;

latLngToCell_Distinct::latLngToCell_Distinct(float *lat_h_, float *lon_h_,
                                             uint64_t *idx_h_, int length_,
                                             int resolution, uint blocks = 64,
                                             uint threads = 32) {
  lat_h = lat_h_;
  lon_h = lon_h_;
  idx_h = idx_h_;
  length = length_;
  size_t byte_float = length * sizeof(float);
  size_t byte_uint64 = length * sizeof(uint64_t);

  gpuErrchk(hipMalloc((float **)&lat_d, byte_float));
  gpuErrchk(hipMalloc((float **)&lon_d, byte_float));
  gpuErrchk(hipMalloc((uint64_t **)&idx_d, byte_uint64));

  gpuErrchk(hipMemcpy(lat_d, lat_h, byte_float, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(lon_d, lon_h, byte_float, hipMemcpyHostToDevice));

  // Blocks, Threads
  kernel_h3<<<blocks, threads>>>(lat_d, lon_d, idx_d, length, resolution);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipPeekAtLastError());

  gpuErrchk(hipMemcpy(idx_h_, idx_d, byte_uint64, hipMemcpyDeviceToHost));

  gpuErrchk(hipFree(lat_d));
  gpuErrchk(hipFree(lon_d));
  gpuErrchk(hipFree(idx_d));
}

latLngToCell_Distinct::~latLngToCell_Distinct() {}

latLngToCell_Unified::latLngToCell_Unified(float *data_h, uint64_t *idx_h,
                                           int length, int resolution,
                                           uint blocks = 64,
                                           uint threads = 32) {
  size_t byte_float = 2 * length * sizeof(float);
  size_t byte_uint64 = length * sizeof(uint64_t);
  float *data_d;
  uint64_t *idx_d;

  gpuErrchk(hipMalloc((float **)&data_d, byte_float));
  gpuErrchk(hipMalloc((uint64_t **)&idx_d, byte_uint64));

  gpuErrchk(hipMemcpy(data_d, data_h, byte_float, hipMemcpyHostToDevice));

  // Blocks, Threads
  kernel_h3_unified<<<blocks, threads>>>(data_d, idx_d, length, resolution);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipPeekAtLastError());

  gpuErrchk(hipMemcpy(idx_h, idx_d, byte_uint64, hipMemcpyDeviceToHost));

  gpuErrchk(hipFree(data_d));
  gpuErrchk(hipFree(idx_d));
}

latLngToCell_Unified::~latLngToCell_Unified() {}